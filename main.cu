#include "hip/hip_runtime.h"
#include <SDL.h>
#include <SDL_image.h>
#include <SDL_ttf.h>
#include <SDL_mixer.h>
#include <iostream>
#include <stdlib.h>  
#include <crtdbg.h>   //for malloc and free
#include <set>
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <vector>
#include <string>
#define _CRTDBG_MAP_ALLOC
#ifdef _DEBUG
#define new new( _NORMAL_BLOCK, __FILE__, __LINE__)
#endif

const int WIDTH = 800;
const int HEIGHT = 600;
SDL_Window* window;
SDL_Renderer* renderer;
bool running;
SDL_Event event;
std::set<std::string> keys;
std::set<std::string> currentKeys;
int mouseX = 0;
int mouseY = 0;
int mouseDeltaX = 0;
int mouseDeltaY = 0;
int mouseScroll = 0;
std::set<int> buttons;
std::set<int> currentButtons;
Uint64 counter = 0;

__device__ struct Pair {
	int16_t x, y;
};
__device__ Pair pairs[4] = { {1, 0}, {0, 1}, {-1, 0}, {0, -1} };

class Pixel {
public:
	uint8_t life = 0;
	uint8_t r = 0, g = 0, b = 0;
	uint16_t x = 0, y = 0;
	void setPos(uint16_t x, uint16_t y) {
		this->x = x;
		this->y = y;
	}
	__device__ void setColor(uint8_t r, uint8_t g, uint8_t b) {
		this->r = r;
		this->g = g;
		this->b = b;
	}
	void draw(Uint32* pixel_ptr, SDL_PixelFormat* format) {
		if (life > 0) {
			pixel_ptr[y * WIDTH + x] = SDL_MapRGB(format, r, g, b);
			life--;
			if (life == 0) {
				r = 0;
				g = 0;
				b = 0;
			}
		}
		else if (r != 0 || g != 0 || b != 0) {
			life = 3;
		}
	}
	__device__ Pair spread(hiprandState* state) {
		Pair dir = pairs[hiprand(state) % 4];

		if (x == 0 && dir.x == -1) {
			dir.x = WIDTH - 1;
		}
		else if (x == WIDTH - 1 && dir.x == 1) {
			dir.x = 0;
		}
		else {
			dir.x += x;
		}

		if (y == 0 && dir.y == -1) {
			dir.y = HEIGHT - 1;
		}
		else if (y == HEIGHT - 1 && dir.y == 1) {
			dir.y = 0;
		}
		else {
			dir.y += y;
		}

		return dir;
	}
};
Pixel pixels[HEIGHT * WIDTH];
Pixel* d_pixels;
size_t p_size = sizeof(Pixel) * size_t(HEIGHT * WIDTH);

SDL_Surface* infoSurface, * saveSurface;
unsigned char* savePixels;


void debug(int line, std::string file) {
	std::cout << "Line " << line << " in file " << file << ": " << SDL_GetError() << std::endl;
}

__global__ void initCurand(unsigned int seed, hiprandState* state) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void spread(Pixel pixels[HEIGHT * WIDTH], hiprandState* state) {
	Pixel* thisPixel = &pixels[threadIdx.x * WIDTH + blockIdx.x];
	if (thisPixel->life > 0) {
		Pair direction = thisPixel->spread(state);
		Pixel* toSpread = &pixels[direction.y * WIDTH + direction.x];
		if (toSpread->life == 0) {
			toSpread->setColor(thisPixel->r + (hiprand(state) % 3 - 1), thisPixel->g + (hiprand(state) % 3 - 1), thisPixel->b + (hiprand(state) % 3 - 1));
		}
	}
}
Uint32 frameStart, spreadStart, drawStart;
int frameTime = 0;
bool timing = true;

int main(int argc, char* argv[]) {
	srand(time(0));
	if (SDL_Init(SDL_INIT_EVERYTHING) == 0 && TTF_Init() == 0 && Mix_OpenAudio(44100, MIX_DEFAULT_FORMAT, 2, 2048) == 0) {
		//Setup
		window = SDL_CreateWindow("Window", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, WIDTH, HEIGHT, 0);
		if (window == NULL) {
			debug(__LINE__, __FILE__);
		}

		renderer = SDL_CreateRenderer(window, -1, 0);
		if (renderer == NULL) {
			debug(__LINE__, __FILE__);
		}

		infoSurface = SDL_GetWindowSurface(window);
		SDL_SetRenderDrawBlendMode(renderer, SDL_BLENDMODE_NONE);
		savePixels = new unsigned char[infoSurface->w * infoSurface->h * infoSurface->format->BytesPerPixel];

		SDL_Texture* texture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGBA8888,
			SDL_TEXTUREACCESS_STREAMING, WIDTH, HEIGHT);
		void* txtPixels;
		int pitch;
		SDL_PixelFormat* format = SDL_AllocFormat(SDL_PIXELFORMAT_RGBA8888);
		Uint32* pixel_ptr;

		for (uint16_t i = 0; i < WIDTH; i++) {
			for (uint16_t j = 0; j < HEIGHT; j++) {
				pixels[j * WIDTH + i].setPos(i, j);
			}
		}

		Pixel* first = &pixels[(HEIGHT + 1) * WIDTH / 2];
		first->r = 127;
		first->g = 127;
		first->b = 127;

		hipSetDevice(0);
		hiprandState* d_state;
		hipMalloc(&d_state, sizeof(hiprandState));
		initCurand << <1, 1 >> > (time(0), d_state);
		hipDeviceSynchronize();
		hipMalloc((void**)&d_pixels, p_size);

		//Main loop
		running = true;
		while (running) {
			frameStart = SDL_GetTicks();
			//handle events
			for (std::string i : keys) {
				currentKeys.erase(i); //make sure only newly pressed keys are in currentKeys
			}
			for (int i : buttons) {
				currentButtons.erase(i); //make sure only newly pressed buttons are in currentButtons
			}
			mouseScroll = 0;
			while (SDL_PollEvent(&event)) {
				switch (event.type) {
				case SDL_QUIT:
					running = false;
					break;
				case SDL_KEYDOWN:
					if (!keys.contains(std::string(SDL_GetKeyName(event.key.keysym.sym)))) {
						currentKeys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym)));
					}
					keys.insert(std::string(SDL_GetKeyName(event.key.keysym.sym))); //add keydown to keys set
					break;
				case SDL_KEYUP:
					keys.erase(std::string(SDL_GetKeyName(event.key.keysym.sym))); //remove keyup from keys set
					break;
				case SDL_MOUSEMOTION:
					mouseX = event.motion.x;
					mouseY = event.motion.y;
					mouseDeltaX = event.motion.xrel;
					mouseDeltaY = event.motion.yrel;
					break;
				case SDL_MOUSEBUTTONDOWN:
					if (!buttons.contains(event.button.button)) {
						currentButtons.insert(event.button.button);
					}
					buttons.insert(event.button.button);
					break;
				case SDL_MOUSEBUTTONUP:
					buttons.erase(event.button.button);
					break;
				case SDL_MOUSEWHEEL:
					mouseScroll = event.wheel.y;
					break;
				}
			}

			spreadStart = SDL_GetTicks();
			hipMemcpy(d_pixels, pixels, p_size, hipMemcpyHostToDevice);
			spread << <WIDTH, HEIGHT >> > (d_pixels, d_state);
			hipDeviceSynchronize();
			hipMemcpy(pixels, d_pixels, p_size, hipMemcpyDeviceToHost);
			if (timing) {
				std::cout << "spread time: " << SDL_GetTicks() - spreadStart;
			}

			drawStart = SDL_GetTicks();
			SDL_LockTexture(texture, NULL, &txtPixels, &pitch);
			pixel_ptr = (Uint32*)txtPixels;
			for (uint16_t i = 0; i < WIDTH; i++) {
				for (uint16_t j = 0; j < HEIGHT; j++) {
					pixels[j* WIDTH + i].draw(pixel_ptr, format);
				}
			}
			SDL_UnlockTexture(texture);
			SDL_RenderCopy(renderer, texture, NULL, NULL);
			SDL_RenderReadPixels(renderer, &infoSurface->clip_rect, infoSurface->format->format, savePixels, infoSurface->w * infoSurface->format->BytesPerPixel);
			SDL_RenderPresent(renderer);
			if (timing) {
				std::cout << " draw time: " << SDL_GetTicks() - drawStart;
			}


			saveSurface = SDL_CreateRGBSurfaceFrom(savePixels, infoSurface->w, infoSurface->h, infoSurface->format->BitsPerPixel, infoSurface->w * infoSurface->format->BytesPerPixel,
				infoSurface->format->Rmask, infoSurface->format->Gmask, infoSurface->format->Bmask, infoSurface->format->Amask);
			//SDL_SaveBMP_RW(saveSurface, SDL_RWFromFile(("Images/Image" + std::to_string(counter) + ".bmp").c_str(), "wb"), 1);
			SDL_FreeSurface(saveSurface);
			counter++;

			frameTime = SDL_GetTicks() - frameStart;
			std::cout << " frame time: " << frameTime << std::endl;
			//hipError_t err = hipGetLastError();
			//std::cout << "Error: " << hipGetErrorString(err) << std::endl;
		}
		//Clean up
		delete[] savePixels;
		SDL_FreeFormat(format);
		hipFree(d_pixels);
		hipFree(d_state);
		if (window) {
			SDL_DestroyWindow(window);
		}
		if (renderer) {
			SDL_DestroyRenderer(renderer);
		}
		TTF_Quit();
		Mix_Quit();
		IMG_Quit();
		SDL_Quit();
		return 0;
	}
	else {
		return 0;
	}
}